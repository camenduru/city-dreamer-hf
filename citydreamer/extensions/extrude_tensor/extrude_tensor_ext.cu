#include "hip/hip_runtime.h"
/**
 * @File:   extrude_tensor_ext.cu
 * @Author: Haozhe Xie
 * @Date:   2023-03-26 11:06:18
 * @Last Modified by: Haozhe Xie
 * @Last Modified at: 2023-05-03 14:55:01
 * @Email:  root@haozhexie.com
 */

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <torch/extension.h>

#define CUDA_NUM_THREADS 512

// Computer the number of threads needed in GPU
inline int get_n_threads(int n) {
  const int pow_2 = std::log(static_cast<float>(n)) / std::log(2.0);
  return max(min(1 << pow_2, CUDA_NUM_THREADS), 1);
}

__global__ void extrude_tensor_ext_cuda_kernel(
    int height, int width, int depth, const int *__restrict__ seg_map,
    const int *__restrict__ height_field, int *__restrict__ volume) {
  int batch_index = blockIdx.x;
  int index = threadIdx.x;
  int stride = blockDim.x;

  seg_map += batch_index * height * width;
  height_field += batch_index * height * width;
  volume += batch_index * height * width * depth;
  for (int i = index; i < height; i += stride) {
    int offset_2d_r = i * width, offset_3d_r = i * width * depth;
    for (int j = 0; j < width; ++j) {
      int offset_2d_c = offset_2d_r + j, offset_3d_c = offset_3d_r + j * depth;
      int seg = seg_map[offset_2d_c];
      int hf = height_field[offset_2d_c];
      for (int k = 0; k < hf + 1; ++k) {
        volume[offset_3d_c + k] = seg;
      }
    }
  }
}

torch::Tensor extrude_tensor_ext_cuda_forward(torch::Tensor seg_map,
                                              torch::Tensor height_field,
                                              int max_height,
                                              hipStream_t stream) {
  int batch_size = seg_map.size(0);
  int height = seg_map.size(2);
  int width = seg_map.size(3);
  torch::Tensor volume = torch::zeros({batch_size, height, width, max_height},
                                      torch::CUDA(torch::kInt32));

  extrude_tensor_ext_cuda_kernel<<<
      batch_size, int(CUDA_NUM_THREADS / CUDA_NUM_THREADS), 0, stream>>>(
      height, width, max_height, seg_map.data_ptr<int>(),
      height_field.data_ptr<int>(), volume.data_ptr<int>());

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in extrude_tensor_ext_cuda_forward: %s\n",
           hipGetErrorString(err));
  }
  return volume;
}
